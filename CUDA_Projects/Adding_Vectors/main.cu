
#include <hip/hip_runtime.h>
#include <stdio.h>

#define SIZE 1024

__global__ 
void kernel(int *a, int *b, int *c, int n)
{
    int i = threadIdx.x;
    // printf("Thread %d\n", i);

    if(i < n)
        c[i] = a[i] + b[i];
}

int main(void)
{
    int *a, *b, *c;

    // Available to both CPU and GPU
    hipMallocManaged(&a, SIZE * sizeof(int));
    hipMallocManaged(&b, SIZE * sizeof(int));
    hipMallocManaged(&c, SIZE * sizeof(int));

    for(int i = 0; i < SIZE; i++) {
        a[i] = i;
        b[i] = i;
        c[i] = 0;
    }

    // Specify the launch configuration of the kernel
    // The first parameter id the number of thread blocks
    // The next parameter is the number of threads within each thread block
    kernel<<<1, SIZE>>> (a, b, c, SIZE);

    // To ensure the CPU waits for the GPU to finish
    hipDeviceSynchronize();

    for(int i = 0; i < 10; i++)
        printf("c[%d] = %d\n", i, c[i]);

    hipFree(a);
    hipFree(b);
    hipFree(b);

    return 0;
}